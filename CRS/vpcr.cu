#include "hip/hip_runtime.h"
#include "vpcr.h"

void VPCR_Init(double *v1, double *v2, double *v3, double *v4, double *v5, double *v6, double *x, double ndata){
  DoubleVecInit(v1,0.0,ndata);
  DoubleVecInit(v2,0.0,ndata);
  DoubleVecInit(v3,0.0,ndata);
  DoubleVecInit(v4,0.0,ndata);
  DoubleVecInit(v5,0.0,ndata);
  DoubleVecInit(v6,0.0,ndata);
  DoubleVecInit(x,0.0,ndata);
}

int VPCR_CRS(double *val, int *col, int *ptr, double *bvec, double *xvec, int ndata, int nnz, double eps, int i_max){
  /* int i, j, k, n; */
  int loop;

  double *rvec, *pvec, *zvec, *Av, *Ap, *x_0, error=0.0;
  double alpha, beta, bnorm, rnorm;
  double zaz, zaz2;

  bool flag=false;
  double t_error=0.0;
  double error_message;

  double st, et, t1;

  FILE *p_x, *p_his;
  p_x=FileInit("./output/VPCR_x.txt", "w");
  p_his=FileInit("./output/VPCR_his.txt", "w");

  st=gettimeofday_sec();

  rvec=Double1Malloc(ndata);
  pvec=Double1Malloc(ndata);
  zvec=Double1Malloc(ndata);
  Av=Double1Malloc(ndata);
  Ap=Double1Malloc(ndata);
  x_0=Double1Malloc(ndata);

  //init vectory
  /* VPCR_Init(rvec, pvec, qvec, svec, zvec, wvec, xvec, ndata); */

  DoubleVecInit(rvec, 0.0, ndata);
  DoubleVecInit(pvec, 0.0, ndata);
  DoubleVecInit(zvec, 0.0, ndata);
  DoubleVecInit(Av, 0.0, ndata);
  DoubleVecInit(Ap, 0.0, ndata);
  DoubleVecInit(xvec, 0.0, ndata);

  DoubleVecCopy(x_0, xvec, ndata);

  // b 2norm
  bnorm = Double2Norm(bvec, ndata);

  //Ax(Av)
  DoubleMVMCSR(Av, val, col, ptr, xvec, ndata);

  //r=b-Ax
  DoubleVecSub(rvec, bvec, Av, ndata);


  //solve z by Az=r
  error_message=InnerSolverSelecter(val, col, ptr, rvec, zvec, ndata, nnz, eps_inner, loop_inner, kskip_inner, fix_inner);
  if(error_message!=0){
    printf("error in vpcr\n");
    return -1;
  }

  //p=z
  DoubleVecCopy(pvec, zvec, ndata);

  //Az(Av)
  DoubleMVMCSR(Av, val, col, ptr, zvec, ndata);

  //Ap=Az
  DoubleVecCopy(Ap, Av, ndata);
  /* //q=s */
  /* DoubleVecCopy(qvec, svec, ndata); */

  // (z,Az)
  zaz=DoubleDot(zvec, Av, ndata); 

  for(loop=0;loop<i_max;loop++){
    //rnorm
    rnorm = Double2Norm(rvec, ndata);
    error=rnorm/bnorm;
    if(verbose){
      printf("Outer %d %.12e\n",loop+1, error);
    }
    fprintf(p_his,"%d %.12e\n",loop+1, error);
    if(error <= eps){
      flag=true;
      break;
    }

    //alpha=(z,Az)/(Ap,Ap)
    alpha = zaz / DoubleDot(Ap, Ap, ndata);

    //x=alpha*pvec+x
    DoubleScalarxpy(xvec, alpha, pvec, xvec, ndata);

    //r=-alpha*Ap+r
    DoubleScalarxpy(rvec, -alpha, Ap, rvec, ndata);

    //init zvec
    DoubleVecInit(zvec, 0.0, ndata);

    //solve z by Az=r
    error_message=InnerSolverSelecter(val, col, ptr, rvec, zvec, ndata, nnz, eps_inner, loop_inner, kskip_inner, fix_inner);
    if(error_message!=0){
      printf("error in vpcr\n");
      return -1;
    }

    //Az
    DoubleMVMCSR(Av, val, col, ptr, zvec, ndata);
    
    //(z,Az)
    zaz2=DoubleDot(zvec, Av, ndata);

    //beta=(z_new,s_new)/(z,s)
    beta = zaz2/zaz;

    zaz=zaz2;

    //p=beta*p+z
    DoubleScalarxpy(pvec, beta, pvec, zvec, ndata);

    //Ap=beta*Ap+Az
    DoubleScalarxpy(Ap, beta, Ap, Av, ndata); 
  } 
  et=gettimeofday_sec();
  t1=et-st;

  FileOutPutVec(p_x, xvec, ndata); 
  t_error=error_check_CRS(val, col, ptr, bvec, xvec, x_0, ndata); 
  printf("|b-ax|2/|b|2=%.1f\n", t_error); 
  printf("Execution Time=%lf s\n", t1);
  
  Double1Free(rvec); 
  Double1Free(pvec); 
  Double1Free(zvec); 
  Double1Free(Av); 
  Double1Free(Ap); 
  Double1Free(x_0); 
  FileClose(p_x); 
  FileClose(p_his); 

  if(flag){ 
    return 1; 
  }
  return 2; 
}
