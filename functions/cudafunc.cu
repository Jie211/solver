#include "cudafunc.h"

void DoubleCudaMalloc(double *ptr, int size){
  hipMalloc((void **)&ptr, sizeof(double)*size);
}
void IntgerCudaMalloc(int *ptr, int size){
  hipMalloc((void **)&ptr, sizeof(int)*size);
}
void DoubleMemCpyH2D(double *host, double *device, int size){
  hipMemcpy(device, host, sizeof(double)*size, hipMemcpyHostToDevice);
}
void IntgerMemCpyH2D(int *host, int *device, int size){
  hipMemcpy(device, host, sizeof(int)*size, hipMemcpyHostToDevice);
}
void DoubleMemCpyD2H(double *host, double *device, int size){
  hipMemcpy(host, device, sizeof(double)*size, hipMemcpyDeviceToHost);
}
void IntgerMemCpyD2H(int *host, int *device, int size){
  hipMemcpy(host, device, sizeof(int)*size, hipMemcpyDeviceToHost);
}
